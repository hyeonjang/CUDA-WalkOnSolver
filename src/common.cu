#include <cuwos/common.h>

int cuda_runtime_version() {
    int version;
    CHECK_CUDA(hipRuntimeGetVersion(&version));
    return version;
}

int cuda_device() {
    int device;
    CHECK_CUDA(hipGetDevice(&device));
    return device;
}

void set_cuda_device(int device) {
    CHECK_CUDA(hipSetDevice(device));
}

bool cuda_supports_virtual_memory(int device) {
    int supports_vmm;
	CHECK_CUDA(hipDeviceGetAttribute(&supports_vmm, CU_DEVICE_ATTRIBUTE_VIRTUAL_ADDRESS_MANAGEMENT_SUPPORTED, device));
	return supports_vmm != 0;
};

size_t cuda_memory_granularity(int device) {
	size_t granularity;
	hipMemAllocationProp prop = {};
	prop.type = hipMemAllocationTypePinned;
	prop.location.type = hipMemLocationTypeDevice;
	prop.location.id = 0;
	hipError_t granularity_result = hipMemGetAllocationGranularity(&granularity, &prop, hipMemAllocationGranularityMinimum);
	if (granularity_result == hipErrorNotSupported) {
		return 1;
	}
	// cu_throw(granularity_result);
	return granularity;
}

MemoryInfo cuda_memory_info() {
    MemoryInfo info;
    CHECK_CUDA(hipMemGetInfo(&info.free, &info.total));
    info.used = info.total - info.free;
    return info;
};